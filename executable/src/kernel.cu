#include "hip/hip_runtime.h"
#include "kernel.hpp"

#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdexcept>

namespace lush::cuda {

    // CUDA Kernel
    __global__ void VecAdd_device(float* A, float* B, float* C) {
        
        // Grab the thread idx for this job
        int i = threadIdx.x;

        // Perform the kernel execution on this thread idx
        C[i] = A[i] + B[i];

    }

    // C++ interface, stl compliant. Could be inlined, but gcc is probably already doing that anyway. We handle all of our host->device work here.
    std::vector<float> VecAdd(int blocks, int threads_per_block, std::vector<float>& A, std::vector<float>& B) {

        // Check an edge case that could cause all sorts of chaos
        if(A.size() != B.size()) {
            throw std::runtime_error("Error: Cannot add vectors of different length. Returning zeros");
            return std::vector<float>(A.size(), 0.0);
        }
        
        // STL->thrust vector type, able to be easily allocated in the GPU
        thrust::device_vector<float> A_device = A;
        float* A_device_ptr = thrust::raw_pointer_cast(A_device.data());

        // STL->thrust vector type, able to be easily allocated in the GPU
        thrust::device_vector<float> B_device = B;
        float* B_device_ptr = thrust::raw_pointer_cast(B_device.data());

        // thrust vector of the size of our parameters that we will dump the output of the kernel to
        thrust::device_vector<float> sum(A_device.size(), 0.0);
        float* sum_device_ptr = thrust::raw_pointer_cast(sum.data());

        // Kernel call. Blocks from pameter, TpB from parameter, pass in pointers to the nice containerized thrust vectors
        VecAdd_device<<<blocks, threads_per_block>>>(A_device_ptr, B_device_ptr, sum_device_ptr);

        // Wait for everone to finish
        hipDeviceSynchronize();

        // Create our STL return type with the same size as the device types. SILENTLY FAILS IF size1 != size2
        std::vector<float> stl_rtn(sum.size());

        // Copy from the device container to the STL host type.
        thrust::copy(sum.begin(), sum.end(), stl_rtn.begin());

        return stl_rtn;
    }

}
